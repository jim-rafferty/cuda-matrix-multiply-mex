#include "hip/hip_runtime.h"
#include "cuda_ops.h"
#include <hipblas.h>

void cublas_matrix_multiply(float *A, float *ImA, float *B, float *ImB, float *C, float *ImC, 
        unsigned int A_m, unsigned int A_n, 
        unsigned int B_m, unsigned int B_n, 
        unsigned int C_m, unsigned int C_n){

    using namespace std;
    // NB: matlab stores complex numbers as separate real and immaginary parts
    // to use the cublas lib we must convert matlabs 2 floats to hipComplex.
    // (which is the same as float2)
    // def complex variables.
    hipComplex *mat_A = new hipComplex[A_m * A_n];
    hipComplex *mat_B = new hipComplex[B_m * B_n];
    hipComplex *mat_C = new hipComplex[C_m * C_n];
    // copy floats to hipComplex
	int i;
	for (i = 0; i < A_m * A_n; i++){;
		mat_A[i].x = A[i]; // real part
        // Im part
        // If the Im part is not present, set it to 0
        // A real matrix will use twice as much memory as nessecary,
        // but it's likely that the input was a double from matlab
        // anyway...
        if (ImA == NULL){
            mat_A[i].y = 0;
        }
        else{
    		mat_A[i].y = ImA[i]; 
        }
	} 
    // repeat operation for matrix B
	for (i = 0; i < B_m * B_n; i++){;
		mat_B[i].x = B[i];
        if (ImB == NULL){
    		mat_B[i].y = 0;
        }
        else{
            mat_B[i].y = ImB[i];
        }
	}

    // def GPU variables
    hipComplex *nv_A;
    hipComplex *nv_B;
    hipComplex *nv_C;

    // allocate mem for GPU vars
    hipMalloc((void **) &nv_A, A_m * A_n * sizeof(hipComplex));
    hipMalloc((void **) &nv_B, B_m * B_n * sizeof(hipComplex));
    hipMalloc((void **) &nv_C, C_m * C_n * sizeof(hipComplex));

    // copy data to GPU
    hipMemcpy(nv_A, mat_A, A_m * A_n * sizeof(hipComplex), hipMemcpyHostToDevice);
    hipMemcpy(nv_B, mat_B, B_m * B_n * sizeof(hipComplex), hipMemcpyHostToDevice);

    hipComplex alf; alf.x = 1; alf.y = 0; 
    hipComplex bet; bet.x = 0; bet.y = 0;
    const hipComplex *alpha = &alf;
    const hipComplex *beta = &bet;
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, A_m, B_n, A_n, alpha, nv_A, A_m, nv_B, A_n, beta, nv_C, A_m);

    // Destroy the handle
    hipblasDestroy(handle);

    // copy solution back
    hipMemcpy(mat_C, nv_C, C_m * C_n * sizeof(hipComplex), hipMemcpyDeviceToHost);

    // copy complex float to separate floats.
	for (i = 0; i < C_m * C_n; i++){;
        C[i] = mat_C[i].x;
        ImC[i] = mat_C[i].y;
	}

    // clean up GPU vars
    hipFree(nv_A);
    hipFree(nv_B);
    hipFree(nv_C);

    // clean up complex vars
    free(mat_A);
    free(mat_B);
    free(mat_C);
}

